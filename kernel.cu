#include "hip/hip_runtime.h"
#include "kernel.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define LAP_3x3 \
{ \
   -1, -1, -1, \
   -1,  8, -1, \
   -1, -1, -1  \
};
#define LAP_5x5 \
{ \
   -1, -3, -4, -3, -1, \
   -3,  0,  6,  0, -3, \
   -4,  6, 20,  6, -4, \
   -3,  0,  6,  0, -3, \
   -1, -3, -4, -3, -1, \
};
#define LAP_7x7 \
{ \
   -2, -3, -4, -6, -4, -3, -2, \
   -3, -5, -4, -3, -4, -5, -3, \
   -4, -4,  9, 20,  9, -4, -4, \
   -6, -3, 20, 36, 20, -3, -6, \
   -4, -4,  9, 20,  9, -4, -4, \
   -3, -5, -4, -3, -4, -5, -3, \
   -2, -3, -4, -6, -4, -3, -2  \
};
const int KERNEL_DIM = 3;
__device__ __constant__ float d_KERNEL[KERNEL_DIM * KERNEL_DIM] = LAP_3x3;

__global__ void laplace (unsigned char* input_image, unsigned char* output_image, int width, int height)
{
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset - x) / width;

    if (offset < width * height)
    {
        float r, g, b;
        r = g = b = 0;

        for (int ox = -(KERNEL_DIM / 2); ox < (KERNEL_DIM / 2) + 1; ox++)
        {
            for (int oy = -(KERNEL_DIM / 2); oy < (KERNEL_DIM / 2) + 1; oy++)
            {
                if ((x + ox) > -1 && (x + ox) < width && (y + oy) > -1 && (y + oy) < height)
                {
                    const int current_offset = (offset + ox + oy * width) * 3;
                    const int kernel_index = ((KERNEL_DIM * KERNEL_DIM) / 2) + ox + (oy * KERNEL_DIM);

                    r += input_image[current_offset]     * d_KERNEL[kernel_index];
                    g += input_image[current_offset + 1] * d_KERNEL[kernel_index];
                    b += input_image[current_offset + 2] * d_KERNEL[kernel_index];
                }
            }
        }
        output_image[offset * 3]     = r;
        output_image[offset * 3 + 1] = g;
        output_image[offset * 3 + 2] = b;
    }
}

void getError (hipError_t err)
{
    if (err != hipSuccess)
    {
        std::cout << "Error " << hipGetErrorString(err) << std::endl;
    }
}

void filter (unsigned char* input_image, unsigned char* output_image, int width, int height)
{
    unsigned char* dev_input;
    unsigned char* dev_output;
    getError(hipMalloc((void**) &dev_input, width*height*3*sizeof(unsigned char)));
    getError(hipMemcpy(dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice));
 
    getError(hipMalloc((void**) &dev_output, width*height*3*sizeof(unsigned char) ));

    dim3 blockDims(512,1,1);
    dim3 gridDims((unsigned int) ceil((double)(width*height/blockDims.x)), 1, 1);

    laplace<<<gridDims, blockDims>>>(dev_input, dev_output, width, height); 

    getError(hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost));

    getError(hipFree(dev_input));
    getError(hipFree(dev_output));
}
